#include "hip/hip_runtime.h"
#include <iostream>
#include <stack>
#include <queue>

#include "src/MyUtils.h"

__global__ void hello_world()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    // 用__global__定义的kernel是异步的，这意味着host不会等待kernel执行完就执行下一步
    hello_world<<<1, 5>>>();

    std::cout << "Hello, World!" << std::endl;
    return 0;
}
