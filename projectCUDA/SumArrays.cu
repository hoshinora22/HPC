#include "hip/hip_runtime.h"
//
// Created by nora1 on 2023/6/10.
//

#include <iostream>
#include <vector>

#include "../include/MyUtils.h"


/// 在cpu上执行向量相加操作
void sumArrays(const double *a, const double *b, double *res, int size)
{
    // 以步长为4进行迭代，每次迭代处理4个double类型的加法
    for (int i = 0; i < size; i += 4)
    {
        res[i] = a[i] + b[i];
        res[i + 1] = a[i + 1] + b[i + 1];
        res[i + 2] = a[i + 2] + b[i + 2];
        res[i + 3] = a[i + 3] + b[i + 3];
    }
}

/// 在gpu上执行向量向量相加操作的 核函数
__global__ void sumArraysGPU(const double *a, const double *b, double *res)
{
    // 每个线程处理一个数组元素，通过索引threadIdx.x访问对应的元素
    auto i = threadIdx.x;
    //auto id = blockIdx.x * blockDim.x + threadIdx.x;
    res[i] = a[i] + b[i];
}

int main()
{
    // 将当前设备设置为第一个设备
    int dev = 0;
    hipSetDevice(dev);

    // 定义了向量的长度
    unsigned long long elementCount = 32;
    printf("vector size : %llu\n", elementCount);

    // 向量所需的存储空间
    auto elementByte = sizeof(double) * elementCount;

    // 在主机上分配内存
    auto *host_a = (double *) malloc(elementByte);
    auto *host_b = (double *) malloc(elementByte);
    auto *host_res = (double *) malloc(elementByte);
    auto *host_resFromGPU = (double *) malloc(elementByte);

    // 将结果向量的内容初始化为0
    memset(host_res, 0, elementByte);
    memset(host_resFromGPU, 0, elementByte);

    // 在GPU上分配内存
    double *dev_a = nullptr;
    double *dev_b = nullptr;
    double *dev_res = nullptr;
    CHECK(hipMalloc((double **) &dev_a, elementByte));
    CHECK(hipMalloc((double **) &dev_b, elementByte));
    CHECK(hipMalloc((double **) &dev_res, elementByte));

    // 初始host化向量的值
    Utils::initialData(host_a, (int) elementByte);
    Utils::initialData(host_b, (int) elementByte);

    // 将host上的向量的数据从主机内存复制到GPU设备内存中
    CHECK(hipMemcpy(dev_a, host_a, elementByte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_b, host_b, elementByte, hipMemcpyHostToDevice));

    // 定义了核函数的执行配置“block”和grid”
    dim3 block(elementCount);
    dim3 grid(elementCount / block.x);

    // 计时器
    double timeStart, timeElapse;
    timeStart = cpuSecond();

    // 执行核函数
    sumArraysGPU<<<grid, block>>>(dev_a, dev_b, dev_res);

    // 加一个同步函数等待核函数执行完毕
    // 如果不加这个同步函数，那么测试的时间是从调用核函数，到核函数返回给主机线程的时间段，而不是核函数的执行时间
    // 加上了同步函数后，计时是从调用核函数开始，到核函数执行完并返回给主机的时间段
    hipDeviceSynchronize();
    timeElapse = cpuSecond() - timeStart;
    printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec\n", grid.x, block.x, timeElapse);

    CHECK(hipMemcpy(host_resFromGPU, dev_res, elementByte, hipMemcpyDeviceToHost));
    sumArrays(host_a, host_b, host_res, (int) elementCount);

    // 数据对比
    Utils::checkResult(host_res, host_resFromGPU, (int)elementCount);

    // GPU上内存释放
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);

    // 主机上的内存释放
    free(host_a);
    free(host_b);
    free(host_res);
    free(host_resFromGPU);

    printf("Fin\n");
    return 0;
}